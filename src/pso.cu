#include "hip/hip_runtime.h"
#include "pso.h"

__global__ void setupCurandInit(hiprandState *state, unsigned long seed)
{
	int id = threadIdx.x;
	hiprand_init(seed, id, 0, &state[id]);
}

__device__ float generateRandom(hiprandState *global_state, int ind)
{
	hiprandState localState = global_state[ind];
	float result = hiprand_uniform(&localState);
	global_state[ind] = localState;
	return result;
}

//initialize the pbest_value
__global__ void initPbestValue(float *pbest_, float *pbest_value_, int num_dimensions_, int num_particles_)
{
	size_t thread_id = threadIdx.x + blockIdx.x * blockDim.x;
	size_t stride = blockDim.x * gridDim.x;
	while (thread_id < num_particles_)
	{
		pbest_value_[thread_id] = fitnessFunction(pbest_ + thread_id * num_dimensions_);
		thread_id += stride;
	}
}

__global__ void updatePbest(float *position_, float *pbest_, float *pbest_value_, int num_dimensions_, int num_particles_)
{
	size_t thread_id = threadIdx.x + blockIdx.x * blockDim.x;
	size_t stride = blockDim.x * gridDim.x;

	while (thread_id < num_particles_)
	{
		float temp_value = fitnessFunction(position_ + num_dimensions_ * thread_id);
		if (temp_value < pbest_value_[thread_id])
		{
			for (int i = 0; i < num_dimensions_; i++)
			{
				pbest_[thread_id * num_dimensions_ + i] = position_[thread_id * num_dimensions_ + i];
			}
			pbest_value_[thread_id] = temp_value;
		}
		thread_id += stride;
	}
}

__global__ void updateParticle(float *position_, float *velocity_, float *pbest_,
							   float *gbest_, float omega_init_, float omega_end_, float c1_, float c2_,
							   float max_velocity_, int num_dimensions_, int num_particles_, int iter,
							   int max_iter_, hiprandState *global_state)
{
	size_t thread_id = threadIdx.x + blockIdx.x * blockDim.x;
	size_t stride = blockDim.x * gridDim.x;

	while (thread_id < num_dimensions_ * num_particles_)
	{
		float omega = (omega_init_ - omega_end_) * (float)(max_iter_ - iter) / (float)max_iter_ + omega_end_;
		velocity_[thread_id] = omega * velocity_[thread_id] + c1_ * generateRandom(global_state, thread_id) * (pbest_[thread_id] - position_[thread_id]) + c2_ * generateRandom(global_state, thread_id + 1) * (gbest_[thread_id % num_dimensions_] - position_[thread_id]);
		if (velocity_[thread_id] > max_velocity_)
			velocity_[thread_id] = max_velocity_;
		position_[thread_id] += velocity_[thread_id];
		thread_id += stride;
	}
}

__device__ __inline__ bool checkMin(float num_a, float num_b)
{
	return num_a < num_b;
}

//function to get gbest
//refer to reduction algorithm
__device__ void warpMin(volatile float *sdata, int blockdim, unsigned int tid, unsigned int thread_id, int num_particles_,  int blocksize)
{
	if (blocksize >= 64 && checkMin(sdata[tid + 32], sdata[tid]) && thread_id + 32 < num_particles_)
	{
		sdata[tid] = sdata[tid + 32];
		sdata[tid + blockdim] = sdata[tid + blockdim + 32];
	}
	if (blocksize >= 32 && checkMin(sdata[tid + 16], sdata[tid]) && thread_id + 16 < num_particles_)
	{
		sdata[tid] = sdata[tid + 16];
		sdata[tid + blockdim] = sdata[tid + blockdim + 16];
	}
	if (blocksize >= 16 && checkMin(sdata[tid + 8], sdata[tid]) && thread_id + 8 < num_particles_)
	{
		sdata[tid] = sdata[tid + 8];
		sdata[tid + blockdim] = sdata[tid + blockdim + 8];
	}
	if (blocksize >= 8 && checkMin(sdata[tid + 4], sdata[tid]) && thread_id + 4 < num_particles_)
	{
		sdata[tid] = sdata[tid + 4];
		sdata[tid + blockdim] = sdata[tid + blockdim + 4];
	}
	if (blocksize >= 4 && checkMin(sdata[tid + 2], sdata[tid]) && thread_id + 2 < num_particles_)
	{
		sdata[tid] = sdata[tid + 2];
		sdata[tid + blockdim] = sdata[tid + blockdim + 2];
	}
	if (blocksize >= 2 && checkMin(sdata[tid + 1], sdata[tid]) && thread_id + 1 < num_particles_)
	{
		sdata[tid] = sdata[tid + 1];
		sdata[tid + blockdim] = sdata[tid + blockdim + 1];
	}
}

__global__ void minGbest(float *pbest_value_, float *pbest_, float *gbest_, int num_particles_, int num_dimensions_, int blocksize)
{
	extern __shared__ float sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int i = blockIdx.x * (blocksize * 2) + tid;
	unsigned int gridSize = blocksize * 2 * gridDim.x;
	if (thread_id > num_particles_)
		return;
	sdata[tid] = pbest_value_[tid];
	sdata[tid + blockDim.x] = i;
	while (i < num_particles_)
	{
		if (checkMin(pbest_value_[i], sdata[tid]))
		{
			sdata[tid] = pbest_value_[i];
			sdata[tid + blockDim.x] = i;
		}

		if (i + blocksize > num_particles_)
			break;

		if (checkMin(pbest_value_[i + blocksize], sdata[tid]))
		{
			sdata[tid] = pbest_value_[i + blocksize];
			sdata[tid + blockDim.x] = i + blocksize;
		}
		i += gridSize;
	}
	__syncthreads();
	if (blocksize >= 1024)
	{
		if (tid < 512 && checkMin(sdata[tid + 512], sdata[tid]) && thread_id + 512 < num_particles_)
		{
			sdata[tid] = sdata[tid + 512];
			sdata[tid + blockDim.x] = sdata[tid + blockDim.x + 512];
		}
		__syncthreads();
	}
	if (blocksize >= 512)
	{
		if (tid < 256 && checkMin(sdata[tid + 256], sdata[tid]) && thread_id + 256 < num_particles_)
		{
			sdata[tid] = sdata[tid + 256];
			sdata[tid + blockDim.x] = sdata[tid + blockDim.x + 256];
		}
		__syncthreads();
	}
	if (blocksize >= 256)
	{
		if (tid < 128 && checkMin(sdata[tid + 128], sdata[tid]) && thread_id + 128 < num_particles_)
		{
			sdata[tid] = sdata[tid + 128];
			sdata[tid + blockDim.x] = sdata[tid + blockDim.x + 128];
		}
		__syncthreads();
	}
	if (blocksize >= 128)
	{
		if (tid < 64 && checkMin(sdata[tid + 64], sdata[tid]) && thread_id + 64 < num_particles_)
		{
			sdata[tid] = sdata[tid + 64];
			sdata[tid + blockDim.x] = sdata[tid + blockDim.x + 64];
		}
		__syncthreads();
	}
	if (tid < 32)
		warpMin(sdata, blockDim.x, tid, thread_id, num_particles_, blocksize);
	if (tid == 0)
	{
		for (int i = 0; i < num_dimensions_; i++)
		{
			gbest_[i] = pbest_[i + (unsigned int)(sdata[blockDim.x]) * num_dimensions_];
		}
		//printf("gbest is %lf",*gbest_);
	}
}

bool PSO::initCuda()
{
	int device_count = 0;
	// Check if CUDA runtime calls work at all
	hipError_t t = hipGetDeviceCount(&device_count);
	if (t != hipSuccess)
	{
		std::cout << "[CUDA] First call to CUDA Runtime API failed. Are the drivers installed?" << std::endl;
		return false;
	}

	// Is there a CUDA device at all?
	checkCudaErrors(hipGetDeviceCount(&device_count));
	if (device_count < 1)
	{
		std::cout << "[CUDA] No CUDA devices found. " << std::endl;
		std::cout << "[CUDA] Make sure CUDA device is powered, connected and available. " << std::endl;
		std::cout << "[CUDA] On laptops: disable powersave/battery mode. " << std::endl;
		std::cout << "[CUDA] Exiting... " << std::endl;
		return false;
	}

	std::cout << "[CUDA] CUDA device(s) found, picking best one " << std::endl;
	std::cout << "[CUDA] " << std::endl;
	// We have at least 1 CUDA device, so now select the fastest (method from Nvidia helper library)
	int device = findCudaDevice(0, 0);

	// Print available device memory
	hipDeviceProp_t properties;
	checkCudaErrors(hipGetDeviceProperties(&properties, device));
	std::cout << "[CUDA] Best device: " << properties.name << std::endl;
	std::cout << "[CUDA] Available global device memory: " << (double)properties.totalGlobalMem / 1024 / 1024 << " MB. " << std::endl;

	// Check compute capability
	if (properties.major < 2)
	{
		std::cout << "[CUDA] Your cuda device has compute capability " << properties.major << properties.minor << ". We need at least 2.0 for atomic operations. Exiting. " << std::endl;
		return false;
	}

	return true;
}

void PSO::getResultCUDA()
{
	if (!initCuda())
		return;
	init();

	hipEvent_t start_vox, stop_vox;
	hiprandState *dev_state;
	checkCudaErrors(hipEventCreate(&start_vox));
	checkCudaErrors(hipEventCreate(&stop_vox));
	float elapsedTime;

	float *dev_position;
	float *dev_velocity;
	float *dev_pbest;
	float *dev_gbest;
	float *dev_pbest_value;

	// Estimate best block and grid size using CUDA Occupancy Calculator
	int block_size_particle;
	int block_size_pbest;
	int min_grid_size_particle;
	int min_grid_size_pbest;
	int grid_size_particle;
	int grid_size_pbest;
	hipOccupancyMaxPotentialBlockSize(&min_grid_size_particle, &block_size_particle, updateParticle, 0, num_dimensions_ * num_particles_);
	hipOccupancyMaxPotentialBlockSize(&min_grid_size_pbest, &block_size_pbest, updatePbest, 0, num_particles_);

	grid_size_particle = (num_particles_ * num_dimensions_ + block_size_particle - 1) / block_size_particle;
	grid_size_pbest = (num_particles_ + block_size_pbest - 1) / block_size_pbest;
	int sdatasize = block_size_pbest * sizeof(float);

	//printPbest();
	//printPbestValue();
	//std::cout<<num_particles_<<" "<<num_dimensions_<<std::endl;
	//std::cout<<block_size_particle<<" "<<block_size_pbest<<" "<<min_grid_size_particle<<" "<<min_grid_size_pbest<<" "<<grid_size_particle<<" "<<grid_size_pbest<<std::endl;

	checkCudaErrors(hipMalloc((void **)&dev_position, sizeof(float) * num_dimensions_ * num_particles_));
	checkCudaErrors(hipMalloc((void **)&dev_velocity, sizeof(float) * num_dimensions_ * num_particles_));
	checkCudaErrors(hipMalloc((void **)&dev_pbest, sizeof(float) * num_dimensions_ * num_particles_));
	checkCudaErrors(hipMalloc((void **)&dev_gbest, sizeof(float) * num_dimensions_));
	checkCudaErrors(hipMalloc((void **)&dev_pbest_value, sizeof(float) * num_particles_));
	checkCudaErrors(hipMalloc(&dev_state, sizeof(float) * num_particles_ * num_dimensions_));

	checkCudaErrors(hipMemcpy(dev_position, position_, sizeof(float) * num_particles_ * num_dimensions_, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_velocity, velocity_, sizeof(float) * num_dimensions_ * num_particles_, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_pbest, pbest_, sizeof(float) * num_dimensions_ * num_particles_, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_gbest, gbest_, sizeof(float) * num_dimensions_, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_pbest_value, pbest_value_, sizeof(float) * num_particles_, hipMemcpyHostToDevice));

	//Optimization
	checkCudaErrors(hipEventRecord(start_vox, 0));
	initPbestValue<<<grid_size_pbest, block_size_pbest>>>(dev_pbest, dev_pbest_value, num_dimensions_, num_particles_);
	hipDeviceSynchronize();
	setupCurandInit<<<grid_size_particle, block_size_particle>>>(dev_state, unsigned(time(NULL)));
	hipDeviceSynchronize();
	for (int i = 0; i < max_iter_; i++)
	{
		updateParticle<<<grid_size_particle, block_size_particle>>>(dev_position, dev_velocity, dev_pbest, dev_gbest, omega_init_, omega_end_, c1_, c2_, max_velocity_, num_dimensions_, num_particles_, 0, max_iter_, dev_state);
		hipDeviceSynchronize();
		updatePbest<<<grid_size_pbest, block_size_pbest>>>(dev_position, dev_pbest, dev_pbest_value, num_dimensions_, num_particles_);
		hipDeviceSynchronize();
		minGbest<<<grid_size_pbest, block_size_pbest, sdatasize * 2>>>(dev_pbest_value, dev_pbest, dev_gbest, num_particles_, num_dimensions_, block_size_pbest);
		hipDeviceSynchronize();
	}

	checkCudaErrors(hipEventRecord(stop_vox, 0));
	checkCudaErrors(hipEventSynchronize(stop_vox));
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start_vox, stop_vox));
	std::cout << "[Time] PSO GPU time: " << elapsedTime << "ms" << std::endl;

	checkCudaErrors(hipMemcpy(position_, dev_position, sizeof(float) * num_particles_ * num_dimensions_, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(velocity_, dev_velocity, sizeof(float) * num_particles_ * num_dimensions_, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pbest_, dev_pbest, sizeof(float) * num_particles_ * num_dimensions_, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(gbest_, dev_gbest, sizeof(float) * num_dimensions_, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(pbest_value_, dev_pbest_value, sizeof(float) * num_particles_, hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(dev_position));
	checkCudaErrors(hipFree(dev_velocity));
	checkCudaErrors(hipFree(dev_pbest));
	checkCudaErrors(hipFree(dev_gbest));
	checkCudaErrors(hipFree(dev_pbest_value));

	// Destroy timers
	checkCudaErrors(hipEventDestroy(start_vox));
	checkCudaErrors(hipEventDestroy(stop_vox));

	//printPbest();
	//printPbestValue();
}
